#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__
#define __HIPCC__
#include <hip/device_functions.h>
#endif

#include <stdio.h>
#include<iostream>

#include "dfa.h"
#include "nfa.h"
#include "nfa_loader.h"
#include "book_loader.h"
#include "infant.h"

__global__ void addi(int* x, int* y, int* z) {

	int i = threadIdx.x;
	z[i] = x[i] + y[i];
	printf("hello gpu thread %d\n", i);

}
void runDFA(dfa* d, char* s, int length) {
	int state = 0;
	for (int i = 0; i < length; i++) {
		printf("current state is %d, reading char %c ", state, s[i]);
		state = d->transitions[state][s[i]];
		printf("moving to %d\n", state);
	}
	if (d->accept[state]) {

		printf("accepting\n");
	}
	else {
		printf("rejecting string %s \n", s);
	}

}


__global__ void DFAGPU(dfa* d, char* s, int length) {
	int state = 0;
	for (int i = 0; i < length; i++) {
		printf("current state is %d, reading char %c ", state, s[i]);
		state = d->transitions[state][s[i]];
		printf("moving to %d\n", state);
	}
	printf("final state %d \n", state);
	if (d->accept[state] != 0) {

		printf("accepting on gpu\n");
	}
	else {
		printf("rejecting string %s on gpu \n", s);
	}

}


void runNFA(nfa* n, char* s, int length) {

	int* active_states = (int*)malloc(NFA_SIZE * sizeof(int));
	memset(active_states, 0, NFA_SIZE * sizeof(int));
	active_states[0] = 1; //start state active

	for (int i = 0; i < length; i++) {

		int* new_states = (int*)malloc(NFA_SIZE * sizeof(int));
		memset(new_states, 0, NFA_SIZE * sizeof(int));
		printf("reading char %c.active states:", s[i]);
		for (int j = 0; j < NFA_SIZE; j++) { //go thru active states
			if (active_states[j] == 1) {
				printf(" %d", j);


				for (int k = 0; k < NFA_SIZE; k++) { //go thru possible transitions
					
					if (n->transitions[j][s[i]][k] == 1) {
						
						new_states[k] = 1;
					}

				}
			}

			

		}
		printf("\n");
		free(active_states);
		active_states = new_states;

	}

	int accepted = 0;
	for (int i = 0; i < NFA_SIZE; i++) {
		if (active_states[i] == 1) {
			printf("state %d active", i);
			if (n->accept[i]) {
				printf(" and accepting");
				accepted = 1;
			}
			printf("\n");
		}
	}
	
	if (accepted == 0) {
		printf("no active states, rejecting %s \n", s);
	}

}
__global__ void runNFAGPU(nfa* n, char* s, int length) {

	int* active_states = (int*)malloc(NFA_SIZE * sizeof(int));
	memset(active_states, 0, NFA_SIZE * sizeof(int));
	active_states[0] = 1; //start state active

	for (int i = 0; i < length; i++) {

		int* new_states = (int*)malloc(NFA_SIZE * sizeof(int));
		memset(new_states, 0, NFA_SIZE * sizeof(int));
		printf("GPU: reading char %c.active states:", s[i]);
		for (int j = 0; j < NFA_SIZE; j++) { //go thru active states
			if (active_states[j] == 1) {
				printf(" %d", j);


				for (int k = 0; k < NFA_SIZE; k++) { //go thru possible transitions

					if (n->transitions[j][s[i]][k] == 1) {

						new_states[k] = 1;
					}

				}
			}



		}
		printf("\n");
		free(active_states);
		active_states = new_states;

	}

	int accepted = 0;
	for (int i = 0; i < NFA_SIZE; i++) {
		if (active_states[i] == 1) {
			printf("GPU state %d active", i);
			if (n->accept[i]) {
				printf(" and accepting");
				accepted = 1;
			}
			printf("\n");
		}
	}

	if (accepted == 0) {
		printf("no active states, rejecting %s \n", s);
	}

}
//USE CUDA TIMERS
//CudaEvent

void launchNFA(nfa* n, char* str, int len, int blocks, int threadsPerBlock, float* memoryTime, float* computationTime)
{
	char* dev_str = nullptr;
	nfa* dev_nfa = nullptr;

	

	hipEvent_t memoryStart, memoryStop; //track memory
	hipEventCreate(&memoryStart);
	hipEventCreate(&memoryStop);


	hipEvent_t computeStart, computeStop; //track compute
	hipEventCreate(&computeStart);
	hipEventCreate(&computeStop);

	hipMalloc((void**)& dev_nfa, 1 * sizeof(nfa)); //allocate device memory
	hipMalloc((void**)& dev_str, len * sizeof(char));
	hipEventRecord(memoryStart); //record start of memory 

	hipMemcpy(dev_nfa, n, 1 * sizeof(nfa), hipMemcpyHostToDevice);
	hipMemcpy(dev_str, str, len * sizeof(char), hipMemcpyHostToDevice);
	
	hipEventRecord(memoryStop); //record end of memory stuff, use event synch to get correct time
	hipEventSynchronize(memoryStop);
	

	//We use event sync instead of device sync because eveny stync will freeze the CPU thread just like device
	// But with the added benefit freezing until the event recording, which is right after the kernel finishes


	hipEventRecord(computeStart); //same procedure for running NFA
	runNFAGPU << <blocks, threadsPerBlock >> > (dev_nfa, dev_str, len);
	hipEventRecord(computeStop);
	hipEventSynchronize(computeStop);
	
	
	hipEventElapsedTime(memoryTime, memoryStart, memoryStop); //see results
	hipEventElapsedTime(computationTime, computeStart, computeStop);
	
	printf("Memory Took: %f ms\n", *memoryTime);
	printf("Computation Took: %f ms\n", *computationTime);
	//clean up
	hipFree(dev_str);
	hipFree(dev_nfa);
}

__global__ void infantAlgorithm(INFANT* nfa, char* book, int bookLength, char* active, char* future, int* acceptCounts) {
	//active and future are both assumed to be nfa state sized


	__shared__ int i;
	i = 0;
	__shared__ char selfLoop[256];
	if (nfa->maxState <= 256)
	{
		for (int i = threadIdx.x; i <= nfa->maxState; i += blockDim.x) {
		
			selfLoop[i] = book[i];
		}
	}


	//for(int i = 0; i < bookLength; i++){
	while (i < bookLength){
	//start in a block given by index and go by block width
		char c = book[i];
		//printf("i is %d according to thread %d and c is %c, bdx is %d \n", i, threadIdx.x,c, blockDim.x);
		//TODO max states
		for (int j = threadIdx.x; j < nfa->maxTransitions[c-FIRST_CHAR]; j += blockDim.x) {
			
			//So here we have 2 state IDs stored together and they're each 16 bits and stored in one 32 bit int
			// the lower 16 are the start and the upper 16 are the end
			// So we get a pointer to that int and then use short pointers to the top and bottom to get the states
			
			
			short* startState;
			short* endState;
			//load as int and instead shift+mask
			/**int* transition = &(nfa->transitions[c-FIRST_CHAR][j]);
			//printf("thread %d is looking at transition %.8x\n", threadIdx.x, *transition);
			
				
				startState = ((short*)transition)+1; //the delights of endianess make you do this at least on my AMD machine
				endState = ((short*)transition);

				int start = (int)(*startState);
				int end = (int)(*endState);**/
				int transition = (nfa->transitions[c - FIRST_CHAR][j]); //use bitshifts to decompose intger into high and low bits
				int start = (transition & 0xFFFF0000) >> 16;
				int end = (transition & 0x0000FFFF);
				//printf("before checking state current is { %d, %d} and future is {%d, %d}\n",(int) active[0], (int)active[1], (int)future[0], (int)future[1]);
				if (active[start] != 0) { //if current state in transition is active then future is active
					future[end] = 1;
					
					
					//printf("in state %d with char %c moving to %d via transition %d in thread %d and i is %d\n", start, c, end, j, threadIdx.x ,i);
					
				}
				//printf("after checking state current is { %d, %d} and future is {%d, %d}\n", (int)active[0], (int)active[1], (int)future[0], (int)future[1]);
			
		}
		

		//make sure future is totally done
		__syncthreads();//copy future to current
		for (int j = threadIdx.x; j <=nfa->maxState; j+= blockDim.x){
			active[j] = future[j];
			//if (nfa->maxState <= 256) {
			//	active[j] = active[j] | selfLoop[j];
			//}
			//else {
				active[j] = active[j] | nfa->selfLoops[j]; //if in self loop continue to run
			//}
			//if (nfa->acceptStates[j] == 1 && active[j] != 0) { //if going to be in accpet state count it
			//	acceptCounts[j] = acceptCounts[j] + 1;
			//	
			//}
			acceptCounts[j] += nfa->acceptStates[j] == 1 && active[j] != 0;
			future[j] = 0;
		}
	
		
		
		//no consistent view between thread blocks
		if (threadIdx.x == 0) {
			i++;

		}
		//make sure threads are on same iteration
		__syncthreads();
	}

	//if (threadIdx.x == 0) {
	//	for (int i = 0; i <= nfa->maxState; i++) {
	//		printf("in state %d, with setting %d\n", i, active[i]);
	//	}
	//	for (int i = 0; i <= nfa->maxState; i++) {
	//		printf("state %d active count is %d\n", i, acceptCounts[i]);
	//	}
	//
	//}



}

void runInfant(INFANT* nfa, char* book, int bookLength, float* memoryTime, float* computationTime, int blocks, int threadsPerBlock) {
	int firsts[NFA_CHARS];
	for (int i = 0; i < NFA_CHARS; i++) {
		firsts[i] = nfa->transitions[i][0]; //copy first transitions to a list of first transitions
	}


	

	char* dev_book = nullptr;
	INFANT* dev_nfa = nullptr;


	int* dev_counts = nullptr; //allocate active counter
	hipMalloc((void**)& dev_counts, nfa->maxState * sizeof(int));
	hipMemset(dev_counts, 0, nfa->maxState * sizeof(int));
	//counts for analysis 
	int* counts = (int*)malloc(nfa->maxState * sizeof(int));

	char current_states[MAX_STATES] = { 0 };
	current_states[0] = 1;

	char* dev_current_states = nullptr; // allocate state array
	hipMalloc((void**)& dev_current_states, MAX_STATES * sizeof(char));
	hipMemcpy(dev_current_states, current_states, MAX_STATES * sizeof(char), hipMemcpyHostToDevice);

	char future_states[MAX_STATES] = { 0 };


	char* dev_future_states = nullptr; // allocate state array
	hipMalloc((void**)& dev_future_states, MAX_STATES * sizeof(char));
	hipMemcpy(dev_future_states, future_states, MAX_STATES * sizeof(char), hipMemcpyHostToDevice);


	hipEvent_t memoryStart, memoryStop; //track memory
	hipEventCreate(&memoryStart);
	hipEventCreate(&memoryStop);


	hipEvent_t computeStart, computeStop; //track compute
	hipEventCreate(&computeStart);
	hipEventCreate(&computeStop);

	hipMalloc((void**)& dev_nfa, 1 * sizeof(INFANT)); //allocate device memory
	hipMalloc((void**)& dev_book, bookLength * sizeof(char));

	hipEventRecord(memoryStart); //record start of memory 

	hipMemcpy(dev_nfa, nfa, 1 * sizeof(INFANT), hipMemcpyHostToDevice);
	hipMemcpy(dev_book, book, bookLength * sizeof(char), hipMemcpyHostToDevice);

	hipEventRecord(memoryStop); //record end of memory stuff, use event synch to get correct time
	hipEventSynchronize(memoryStop);


	//We use event sync instead of device sync because eveny sync will freeze the CPU thread just like device
	// But with the added benefit freezing until the event recording, which is right after the kernel finishes


	hipEventRecord(computeStart); //same procedure for running NFA
	//runNFAGPU << <blocks, threadsPerBlock >> > (dev_nfa, dev_str, len);
	infantAlgorithm << <blocks, threadsPerBlock >> > (dev_nfa, dev_book, bookLength, dev_current_states, dev_future_states, dev_counts);
	hipEventRecord(computeStop);
	hipEventSynchronize(computeStop);


	hipEventElapsedTime(memoryTime, memoryStart, memoryStop); //see results
	hipEventElapsedTime(computationTime, computeStart, computeStop);

	printf("Memory Took: %f ms\n", *memoryTime);
	printf("Computation Took: %f ms\n", *computationTime);

	hipMemcpy(counts, dev_counts, nfa->maxState * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < nfa->maxState; i++) {
		printf("state %d count is %d\n", i, counts[i]);
	}

	//clean up
	hipFree(dev_book);
	hipFree(dev_nfa);
	hipFree(dev_current_states);
	hipFree(dev_future_states);


}



void runExperiment(char* book_title, iNFAnt* automaton) {

	std::string s = loadBook(book_title);
	std::string* s2 = &s;
	int char_count;
	char* book = processBook(s2, &char_count);

	float memoryTime;
	float computationTime;
	char* st = "romeo and juliet died";
	//runInfant(nfa2, st, strlen(st), &memoryTime, &computationTime);
	runInfant(automaton, book, char_count, &memoryTime, &computationTime, 1, 21);
	printf("book is %d long\n", char_count);

}

int main()
{

	//std::string s = loadBook("D:/CUDFA/CUDFA/x64/Debug/romeo_and_juliet.txt");
	//std::string* s2 = &s;
	//int char_count;
	//char* book = processBook(s2, &char_count);

	//iNFAnt* nfa = getiNFAnt();
	//
	//addTransition(nfa, 'a', 0, 0); //loop from 0 
	//addTransition(nfa, 'a', 1, 0); //from accept states move to reject when you see a
	//addTransition(nfa, 'a', 2, 0); //from accept states move to reject when you see a
	//
	//
	//
	//addTransition(nfa, 'b', 1, 1); //jump between accept states 
	//addTransition(nfa, 'b', 0, 1); //move to accept from sta
	//addTransition(nfa, 'b', 2, 1); //jump between accept states
	//
	//addTransition(nfa, 'c', 2, 2); //jump between accept states 
	//addTransition(nfa, 'c', 0, 2); //move to accept from sta
	//addTransition(nfa, 'c', 1, 2); //jump between accept states
	//
	//
	////char* str = "abaaaabc";
	//
	//iNFAnt* nfa2 = getiNFAnt();
	//
	////romeos goes from 0 to 5
	////addTransition(nfa2, 'r', 0, 1);
	////addTransition(nfa2, 'o', 1, 2);
	////addTransition(nfa2, 'm', 2, 3);
	////addTransition(nfa2, 'e', 3, 4);
	////addTransition(nfa2, 'o', 4, 5);
	//addString(nfa2, "romeo", 0);
	//addTransition(nfa2, 'R', 0, 1);
	//
	//addTransition(nfa2, 'j', 0, 6);
	//addString(nfa2, "uliet", 6);
	////addTransition(nfa2, 'u', 6, 7);
	////addTransition(nfa2, 'l', 7, 8);
	////addTransition(nfa2, 'i', 8, 9);
	////addTransition(nfa2, 'e', 9, 10);
	////addTransition(nfa2, 't', 10, 11);
	//addTransition(nfa2, 'J', 0, 6);
	//addEpsilon(nfa2, 0, 12);
	//
	//int maxState = nfa2->maxState;
	//
	//addTransition(nfa2, 'C', 0, maxState+1);
	//addString(nfa2, "apulet", maxState+1); //int check for max state
	//maxState = nfa2->maxState;
	//nfa2->acceptStates[maxState-1] = 1; //accept the string capulet
	//
	//nfa2->selfLoops[0] = 1; // self loop in first state
	//
	////now look for the word "the" followed by 1...10
	//
	//
	//nfa2->acceptStates[5] = 1;
	//nfa2->acceptStates[11]  = 1;
	//nfa2->acceptStates[12] = 1;
	//
	//maxState = nfa2->maxState;
	//int groupOfManyStart = maxState;
	//addEpsilon(nfa2, 0, groupOfManyStart);
	//addEpsilonString(nfa2, groupOfManyStart, 3);
	//maxState = nfa2->maxState;
	//int groupof3 = nfa2->maxState - 1;
	//
	//
	//nfa2->acceptStates[maxState-1] = 1;
	//
	//
	//
	//
	iNFAnt* experimentalNFA = getiNFAnt();
	addEpsilon(experimentalNFA, 0, 0);//always loop beginning 

	addString(experimentalNFA, "romeo", 0); //Look for romeo in all books, helps as debuggin sanity check
	addTransition(experimentalNFA, 'R', 0, 1); //capital
	
	int romeoAccept = experimentalNFA->maxState;
	experimentalNFA->acceptStates[romeoAccept] = 1; //accept romeo
	//experimentalNFA->acceptStates[romeoAccept + 1] = 1; //count chars (6)
	std::cout << "romeo accept state is " << romeoAccept << std::endl;
	addTransition(experimentalNFA, 'O', 0, 7); //Now try OF THE
	addTransition(experimentalNFA, 'o', 0, 7);
	addString(experimentalNFA, "f the",7); //search for the string "of the"
	int ofTheAccept = experimentalNFA->maxState;
	experimentalNFA->acceptStates[ofTheAccept] = 1;
	std::cout << "of the accept state is " << ofTheAccept << std::endl;
	
	
	
	experimentalNFA->maxState++;


	runExperiment("D:/CUDFA/CUDFA/x64/Debug/romeo_and_juliet.txt", experimentalNFA);
	getchar();
	return 0;

	/**
	int x[3] = { 1,2,3 };
	int y[3] = { 4,5,6 };
	int z[3] = { 0 };

	int* dev_z = nullptr;
	int* dev_x = nullptr;
	int* dev_y = nullptr;

	hipMalloc((void**)& dev_x, 3 * sizeof(int));
	hipMalloc((void**)& dev_y, 3 * sizeof(int));
	hipMalloc((void**)& dev_z, 3 * sizeof(int));


	printf("%d\n", hipGetLastError());
	hipMemcpy(dev_x, x, 3 * sizeof(int), hipMemcpyHostToDevice);
	printf("%d\n", hipGetLastError());
	hipMemcpy(dev_y, y, 3 * sizeof(int), hipMemcpyHostToDevice);

	printf("%d\n", hipGetLastError());
	addi << <1, 3 >> > (dev_x, dev_y, dev_z);
	printf("%d\n", hipGetLastError());
	hipDeviceSynchronize();
	hipMemcpy(&z, dev_z, 3 * sizeof(int), hipMemcpyDeviceToHost);
	printf("%d\n", hipGetLastError());
	for (int i = 0; i < 3; i++) {
		printf("%d\n", z[i]);
	}


	//dfa* d = (dfa*)malloc(1 * sizeof(dfa)); //alocate dfa and make sure its all -1
	//memset(d, -1, 1 * sizeof(dfa)); //make accept states not true
	//memset(d->accept, 0, DFA_SIZE * sizeof(int));
	//dfa* d = makeDFA();
	//d->accept[1] = 1;
	//d->transitions[0]['a'] = 1;// chars are numbers, very hacky
	//d->transitions[0]['b'] = 0;
	//d->transitions[1]['a'] = 1;
	//d->transitions[1]['b'] = 0; //*a
	char* str = "baab";
	//runDFA(d, str, strlen(str));
	//
	//
	//dfa* dev_dfa = nullptr;
	//char* dev_str = nullptr;
	//
	//hipMalloc((void**)& dev_dfa, 1 * sizeof(dfa));
	//hipMalloc((void**)& dev_str, strlen(str) * sizeof(char));
	//
	//hipMemcpy(dev_dfa, d, 1 * sizeof(dfa), hipMemcpyHostToDevice);
	//hipMemcpy(dev_str, str, strlen(str) * sizeof(char), hipMemcpyHostToDevice);
	//DFAGPU << <1, 1 >> > (dev_dfa, dev_str, strlen(str));
	//hipDeviceSynchronize();


	
	nfa* n = makeNFA();
	n->accept[1] = 1;
	n->accept[2] = 1;
	n->transitions[0]['a'][1] = 1; //from state 0 when it sees a go to state 1 or 2
	n->transitions[0]['a'][2] = 1;

	n->transitions[1]['a'][1] = 1; //when in accept state, stay there 
	n->transitions[1]['a'][2] = 1;
	n->transitions[2]['a'][1] = 1;
	n->transitions[2]['a'][2] = 1;

	n->transitions[0]['b'][0] = 1;//when theres a b always go to a reject state
	n->transitions[1]['b'][0] = 1;
	n->transitions[2]['b'][0] = 1;

	runNFA(n, str, strlen(str));
	float memoryTime = 1;
	float computeTime;
	launchNFA(n, str, strlen(str), 1, 1, &memoryTime, &computeTime); */

	
	
}






